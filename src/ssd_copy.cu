#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, Scientific Computing and Imaging Institute, University of Utah
 *
 * See COPYRIGHT in top-level directory.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "ss_data.h"
#include "dspaces-common.h"

#define CUDA_ASSERT(x)                                                          \
    do                                                                          \
        {                                                                       \
            if (!(x))                                                           \
                {                                                               \
                    fprintf(stderr, "%s, line %i (%s):"                \
                            "Assertion %s failed!\n",                           \
                            __FILE__, __LINE__, __func__, #x);    \
                    return dspaces_ERR_CUDA;                                    \
                }                                                               \
        } while (0)

#define CUDA_ASSERT_RT(stmt)				                                        \
    do                                                                          \
        {                                                                       \
            hipError_t err = (stmt);                                           \
            if (err != hipSuccess) {                                           \
                fprintf(stderr, "%s, line %i (%s):"                    \
                        "%s failed, Err Code: (%s)\n",                          \
                        __FILE__, __LINE__, __func__, #stmt,      \
                        hipGetErrorString(err));                               \
            }                                                                   \
            CUDA_ASSERT(hipSuccess == err);                                \
        } while (0)

__global__ void copy_subarray_c_double(double *dst, double *src, int dst_nx, int dst_ny, int dst_nz,
    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i * dst_ny * dst_nz + j * dst_nz + k] = src[i * src_ny * src_nz + j * src_nz + k];
    }
}

__global__ void copy_subarray_c_float(float *dst, float *src, int dst_nx, int dst_ny, int dst_nz,
    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i * dst_ny * dst_nz + j * dst_nz + k] = src[i * src_ny * src_nz + j * src_nz + k];
    }
}

__global__ void copy_subarray_c_short(short *dst, short *src, int dst_nx, int dst_ny, int dst_nz,
    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i * dst_ny * dst_nz + j * dst_nz + k] = src[i * src_ny * src_nz + j * src_nz + k];
    }
}

__global__ void copy_subarray_c_char(char *dst, char *src, int dst_nx, int dst_ny, int dst_nz,
    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i * dst_ny * dst_nz + j * dst_nz + k] = src[i * src_ny * src_nz + j * src_nz + k];
    }
}

__global__ void copy_subarray_f_double(double *dst, double *src, int dst_nx, int dst_ny, int dst_nz,
                                    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i + j * dst_nx + k * dst_nx * dst_ny] = src[i + j * src_nx + k * src_nx * src_ny];
    }
}

__global__ void copy_subarray_f_float(float *dst, float *src, int dst_nx, int dst_ny, int dst_nz,
    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i + j * dst_nx + k * dst_nx * dst_ny] = src[i + j * src_nx + k * src_nx * src_ny];
    }
}

__global__ void copy_subarray_f_short(short *dst, short *src, int dst_nx, int dst_ny, int dst_nz,
    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i + j * dst_nx + k * dst_nx * dst_ny] = src[i + j * src_nx + k * src_nx * src_ny];
    }
}

__global__ void copy_subarray_f_char(char *dst, char *src, int dst_nx, int dst_ny, int dst_nz,
    int src_nx, int src_ny, int src_nz, int sub_nx, int sub_ny, int sub_nz)
{
    //==============================================================================
    // 2 Registers | 3 arguments
    //==============================================================================
    int   i, j, k;
    //==============================================================================

    // Identify current thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    k = blockIdx.z * blockDim.z + threadIdx.z;

    if ( (i < sub_nx) && (j < sub_ny) && (k < sub_nz) ) {
        dst[i + j * dst_nx + k * dst_nx * dst_ny] = src[i + j * src_nx + k * src_nx * src_ny];
    }
}

extern "C" int matrix_copy_cuda_f_double(struct matrix *dst, struct matrix *src)
{
    double *d = (double*) dst->pdata;
    double *s = (double*) src->pdata;

    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    // Use non-parallel design for unit benchmark
    hipStream_t stream;
    CUDA_ASSERT_RT(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[0];
        src_off = src->mat_view.lb[0];
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0];
        src_off = src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0];
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1;

        dst_off3 = (dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0];
        src_off3 = (src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0];

        dst_stride3 = dst->dist[2] * dst->dist[1] * dst->dist[0];
        src_stride3 = src->dist[2] * dst->dist[1] * dst->dist[0];

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_double<<<dimgrid, dimblock, 0, stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                CUDA_ASSERT_RT(hipStreamSynchronize(stream));
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}

extern "C" int matrix_copy_cuda_f_float(struct matrix *dst, struct matrix *src)
{
    float *d = (float*) dst->pdata;
    float *s = (float*) src->pdata;

    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    // Use non-parallel design for unit benchmark
    hipStream_t stream;
    CUDA_ASSERT_RT(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[0];
        src_off = src->mat_view.lb[0];
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0];
        src_off = src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0];
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1;

        dst_off3 = (dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0];
        src_off3 = (src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0];

        dst_stride3 = dst->dist[2] * dst->dist[1] * dst->dist[0];
        src_stride3 = src->dist[2] * dst->dist[1] * dst->dist[0];

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_float<<<dimgrid, dimblock, 0, stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                CUDA_ASSERT_RT(hipStreamSynchronize(stream));
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}

extern "C" int matrix_copy_cuda_f_short(struct matrix *dst, struct matrix *src)
{
    short *d = (short*) dst->pdata;
    short *s = (short*) src->pdata;

    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    // Use non-parallel design for unit benchmark
    hipStream_t stream;
    CUDA_ASSERT_RT(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[0];
        src_off = src->mat_view.lb[0];
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0];
        src_off = src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0];
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1;

        dst_off3 = (dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0];
        src_off3 = (src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0];

        dst_stride3 = dst->dist[2] * dst->dist[1] * dst->dist[0];
        src_stride3 = src->dist[2] * dst->dist[1] * dst->dist[0];

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_short<<<dimgrid, dimblock, 0, stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                CUDA_ASSERT_RT(hipStreamSynchronize(stream));
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}

extern "C" int matrix_copy_cuda_f_char(struct matrix *dst, struct matrix *src)
{
    char *d = (char*) dst->pdata;
    char *s = (char*) src->pdata;
    
    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    // Use non-parallel design for unit benchmark
    hipStream_t stream;
    CUDA_ASSERT_RT(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    // char function is used for arbitrary data types
    // Therefore, it needs to multiply elem_size to calculate the offsets and copy sizes
    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = (dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1) * dst->size_elem;
        sub_ny = 1 * dst->size_elem;
        sub_nz = 1 * dst->size_elem;
        dst_off = dst->mat_view.lb[0] * dst->size_elem;
        src_off = src->mat_view.lb[0] * dst->size_elem;
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = (dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1) * dst->size_elem;
        sub_ny = (dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1) * dst->size_elem;
        sub_nz = 1 * dst->size_elem;
        dst_off = (dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0]) * dst->size_elem;
        src_off = (src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0]) * dst->size_elem;
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = (dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1) * dst->size_elem;
        sub_ny = (dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1) * dst->size_elem;
        sub_nz = (dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1) * dst->size_elem;

        dst_off3 = ((dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0]) * dst->size_elem;
        src_off3 = ((src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0]) * dst->size_elem;

        dst_stride3 = (dst->dist[2] * dst->dist[1] * dst->dist[0]) * dst->size_elem;
        src_stride3 = (src->dist[2] * dst->dist[1] * dst->dist[0]) * dst->size_elem;

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_char<<<dimgrid, dimblock, 0, stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                CUDA_ASSERT_RT(hipStreamSynchronize(stream));
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}

extern "C" int matrix_copy_cuda_f_double_async(struct matrix *dst, struct matrix *src, hipStream_t *stream)
{
    double *d = (double*) dst->pdata;
    double *s = (double*) src->pdata;

    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[0];
        src_off = src->mat_view.lb[0];
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0];
        src_off = src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0];
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1;

        dst_off3 = (dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0];
        src_off3 = (src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0];

        dst_stride3 = dst->dist[2] * dst->dist[1] * dst->dist[0];
        src_stride3 = src->dist[2] * dst->dist[1] * dst->dist[0];

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_double<<<dimgrid, dimblock, 0, *stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}

extern "C" int matrix_copy_cuda_f_float_async(struct matrix *dst, struct matrix *src, hipStream_t *stream)
{
    float *d = (float*) dst->pdata;
    float *s = (float*) src->pdata;

    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[0];
        src_off = src->mat_view.lb[0];
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0];
        src_off = src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0];
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1;

        dst_off3 = (dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0];
        src_off3 = (src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0];

        dst_stride3 = dst->dist[2] * dst->dist[1] * dst->dist[0];
        src_stride3 = src->dist[2] * dst->dist[1] * dst->dist[0];

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_float<<<dimgrid, dimblock, 0, *stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}

extern "C" int matrix_copy_cuda_f_short_async(struct matrix *dst, struct matrix *src, hipStream_t *stream)
{
    short *d = (short*) dst->pdata;
    short *s = (short*) src->pdata;

    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[0];
        src_off = src->mat_view.lb[0];
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = 1;
        dst_off = dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0];
        src_off = src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0];
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1;
        sub_ny = dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1;
        sub_nz = dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1;

        dst_off3 = (dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0];
        src_off3 = (src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0];

        dst_stride3 = dst->dist[2] * dst->dist[1] * dst->dist[0];
        src_stride3 = src->dist[2] * dst->dist[1] * dst->dist[0];

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_short<<<dimgrid, dimblock, 0, *stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}

extern "C" int matrix_copy_cuda_f_char_async(struct matrix *dst, struct matrix *src, hipStream_t *stream)
{
    char *d = (char*) dst->pdata;
    char *s = (char*) src->pdata;
    
    // int BLOCK_THREAD_SIZE = 1024;
    int BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z;
    int sub_nx, sub_ny, sub_nz;
    int GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z;

    uint64_t dst9, dst8, dst7, dst6, dst5, dst4, dst3;
    uint64_t dst_off9 = 0, dst_off8 = 0, dst_off7 = 0,
             dst_off6 = 0, dst_off5 = 0, dst_off4 = 0;
    uint64_t src9, src8, src7, src6, src5, src4, src3;
    uint64_t src_off9 = 0, src_off8 = 0, src_off7 = 0,
             src_off6 = 0, src_off5 = 0, src_off4 = 0;
    uint64_t dst_off3, src_off3;
    uint64_t dst_off, src_off; 
    uint64_t dst_stride3, src_stride3;

    // char function is used for arbitrary data types
    // Therefore, it needs to multiply elem_size to calculate the offsets and copy sizes
    if(dst->num_dims == 1) {
        BLOCK_SIZE_X = 1024; 
        BLOCK_SIZE_Y = 1;
        BLOCK_SIZE_Z = 1;
        sub_nx = (dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1) * dst->size_elem;
        sub_ny = 1 * dst->size_elem;
        sub_nz = 1 * dst->size_elem;
        dst_off = dst->mat_view.lb[0] * dst->size_elem;
        src_off = src->mat_view.lb[0] * dst->size_elem;
    } else if(dst->num_dims == 2) {
        BLOCK_SIZE_X = 32;
        BLOCK_SIZE_Y = 32;
        BLOCK_SIZE_Z = 1;
        sub_nx = (dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1) * dst->size_elem;
        sub_ny = (dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1) * dst->size_elem;
        sub_nz = 1 * dst->size_elem;
        dst_off = (dst->mat_view.lb[1] * dst->dist[0] + dst->mat_view.lb[0]) * dst->size_elem;
        src_off = (src->mat_view.lb[1] * src->dist[0] + src->mat_view.lb[0]) * dst->size_elem;
    } else { 
        // ndims >= 3 will use 3D kernel in loops, so the params are the same
        BLOCK_SIZE_X = 16;
        BLOCK_SIZE_Y = 8;
        BLOCK_SIZE_Z = 8;
        sub_nx = (dst->mat_view.ub[0] - dst->mat_view.lb[0] + 1) * dst->size_elem;
        sub_ny = (dst->mat_view.ub[1] - dst->mat_view.lb[1] + 1) * dst->size_elem;
        sub_nz = (dst->mat_view.ub[2] - dst->mat_view.lb[2] + 1) * dst->size_elem;

        dst_off3 = ((dst->mat_view.lb[2] * dst->dist[1] + dst->mat_view.lb[1]) * dst->dist[0] + dst->mat_view.lb[0]) * dst->size_elem;
        src_off3 = ((src->mat_view.lb[2] * src->dist[1] + src->mat_view.lb[1]) * src->dist[0] + src->mat_view.lb[0]) * dst->size_elem;

        dst_stride3 = (dst->dist[2] * dst->dist[1] * dst->dist[0]) * dst->size_elem;
        src_stride3 = (src->dist[2] * dst->dist[1] * dst->dist[0]) * dst->size_elem;

        // only ndims == 3 use fixed dst & src offset, others will change the values as excuted in loops
        if(dst->num_dims ==3) {
            dst_off = dst_off3;
            src_off = src_off3;
        }
    }

    GRID_SIZE_X = (sub_nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    GRID_SIZE_Y = (sub_ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    GRID_SIZE_Z = (sub_nz + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z;
    
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z );
    dim3 dimgrid( GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z );

    switch(dst->num_dims) {
        case 1:
            goto ndimleq3;
            break;
        case 2:
            goto ndimleq3;
            break;
        case 3:
            goto ndimleq3;
            break;
        case 4:
            goto ndim4;
            break;
        case 5:
            goto ndim5;
            break;
        case 6:
            goto ndim6;
            break;
        case 7:
            goto ndim7;
            break;
        case 8:
            goto ndim8;
            break;
        case 9:
            goto ndim9;
            break;
        case 10:
            goto ndim10;
            break;
        default:
            return dspaces_ERR_INVALID_ARG;
            break;
        }

ndim10:
    for(dst9 = dst->mat_view.lb[9], src9 = src->mat_view.lb[9];
        dst9 <= dst->mat_view.ub[9]; dst9++, src9++) {
        dst_off9 = dst9 * dst->dist[8];
        src_off9 = src9 * src->dist[8];
    ndim9:
        for(dst8 = dst->mat_view.lb[8], src8 = src->mat_view.lb[8];
            dst8 <= dst->mat_view.ub[8]; dst8++, src8++) {
            dst_off8 = (dst_off9 + dst8) * dst->dist[7];
            src_off8 = (src_off9 + src8) * src->dist[7];
        ndim8:
            for(dst7 = dst->mat_view.lb[7], src7 = src->mat_view.lb[7];
                dst7 <= dst->mat_view.ub[7]; dst7++, src7++) {
                dst_off7 = (dst_off8 + dst7) * dst->dist[6];
                src_off7 = (src_off8 + src7) * src->dist[6];
            ndim7:
                for(dst6 = dst->mat_view.lb[6], src6 = src->mat_view.lb[6];
                    dst6 <= dst->mat_view.ub[6]; dst6++, src6++) {
                    dst_off6 = (dst_off7 + dst6) * dst->dist[5];
                    src_off6 = (src_off7 + src6) * src->dist[5];
                ndim6:
                    for(dst5 = dst->mat_view.lb[5], src5 = src->mat_view.lb[5];
                        dst5 <= dst->mat_view.ub[5]; dst5++, src5++) {
                        dst_off5 = (dst_off6 + dst5) * dst->dist[4];
                        src_off5 = (src_off6 + src5) * src->dist[4];
                    ndim5:
                        for(dst4 = dst->mat_view.lb[4], src4 = src->mat_view.lb[4];
                            dst4 <= dst->mat_view.ub[4]; dst4++, src4++) {
                            dst_off4 = (dst_off5 + dst4) * dst->dist[3];
                            src_off4 = (src_off5 + src4) * src->dist[3];
                        ndim4:
                            for(dst3 = dst->mat_view.lb[3], src3 = src->mat_view.lb[3];
                                dst3 <= dst->mat_view.ub[3]; dst3++, src3++) {
                                dst_off = (dst_off4 + dst3) * dst_stride3 + dst_off3;
                                src_off = (src_off4 + src3) * src_stride3 + dst_off3;
                            ndimleq3:
                                copy_subarray_f_char<<<dimgrid, dimblock, 0, *stream>>>(&d[dst_off],
                                    &s[src_off], dst->dist[0], dst->dist[1], dst->dist[2],
                                    src->dist[0], src->dist[1], src->dist[2], sub_nx, sub_ny, sub_nz);
                                if(src->num_dims <= 3)
                                    return dspaces_SUCCESS;
                            }
                            if(src->num_dims == 4)
                                return dspaces_SUCCESS;
                        }
                        if(src->num_dims == 5)
                            return dspaces_SUCCESS;
                    }
                    if(src->num_dims == 6)
                        return dspaces_SUCCESS;
                }
                if(src->num_dims == 7)
                    return dspaces_SUCCESS;
            }
            if(src->num_dims == 8)
                return dspaces_SUCCESS;
        }
        if(src->num_dims == 9)
            return dspaces_SUCCESS;
    }
    return dspaces_SUCCESS;
}
